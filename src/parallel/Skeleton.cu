#include "hip/hip_runtime.h"
//
//  Image Skeletonisation - Copyright (c) 2014 Marco Antognini <antognini.marco@gmail.com>
//  Under zlib/png license. Refer to LICENSE for the full text.
//

#include "Image.cuh"
#include "Skeleton.hpp"

#include <SFML/System/Clock.hpp>

#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform_reduce.h>

#ifdef DEBUG
#include <iostream>
#define DEBUG_PRINT(x) (x)
#else
#define DEBUG_PRINT(x)
#endif


// Run one full iteration of the thinning algorithm.
// Both buffer1 and buffer2 at the end contain the same data.
bool erode(BWImage<DevicePixels>& buffer1, BWImage<DevicePixels>& buffer2);

// Run one sub-cycle of the thinning algorithm.
// Buffer1 is the source and buffer2 will be thinned if appropriate.
bool erodeIter(BWImage<DevicePixels> const& buffer1, BWImage<DevicePixels>& buffer2, bool firstSubiter);

//************************************************************************************************//
//                                      IMPLEMENTATION                                            //
//************************************************************************************************//

void skeleton(std::string const& input, std::string const& output, sf::Time& elapsed, int& iterations)
{
    // Load the image from disk
    sf::Clock clk;
    BWImage<HostPixels> hostImg = loadImage(input);

#ifndef ENABLE_SAVE_IMAGE
    // Ignore I/O time when not saving the image
    clk.restart();
#endif

    // Create two buffers for the thinning algorithm
    // We do it here to avoid reallocating device memory in a loop
    // The first one contains
    BWImage<DevicePixels> buffer1(hostImg.width, hostImg.height);
    buffer1.copyFrom(hostImg); // Copy from host to device

    BWImage<DevicePixels> buffer2(hostImg.width, hostImg.height);
    buffer2.copyFrom(buffer1); // Copy from device to device

    iterations = 0;
    for (bool run = true; run; ++iterations) {
        // buffer1 and buffer2 will be modified with the new image
        DEBUG_PRINT(std::cout << "thinning....\n");
        run = erode(buffer1, buffer2);
    }

    // Fetch the skeleton from the device
    hostImg.copyFrom(buffer1);

    // Save the skeleton back image
#ifdef ENABLE_SAVE_IMAGE
    saveImage(output, hostImg);
#endif

    // Included I/O time when ENABLE_SAVE_IMAGE is defined
    elapsed = clk.restart();
}

bool erode(BWImage<DevicePixels>& buffer1, BWImage<DevicePixels>& buffer2)
{
    DEBUG_PRINT(std::cout << "\t1st subcycle....\n");
    bool const eroded1 = erodeIter(buffer1, buffer2, true);

    DEBUG_PRINT(std::cout << "\tcopy data......\n");
    buffer1.copyFrom(buffer2); // device to device

    DEBUG_PRINT(std::cout << "\t2nd subcycle....\n");
    bool const eroded2 = erodeIter(buffer1, buffer2, false);

    DEBUG_PRINT(std::cout << "\tcopy data......\n");
    buffer1.copyFrom(buffer2); // device to device

    DEBUG_PRINT(std::cout << "\tdone\n");

    return eroded1 or eroded2;
}

struct ErodePixelFunctor : thrust::unary_function<int, bool>
{
    bool const firstSubiter;    // Flag for sub-iteration identity
    Pixel const* pbuffer1;      // Raw pointer to the device memory of size width * height
    Pixel* pbuffer2;            // Idem
    int const width;    // Image width
    int const height;   // Image height

    ErodePixelFunctor(bool firstSubiter,
                      Pixel const* pbuffer1, Pixel* pbuffer2,
                      int width, int height)
        : firstSubiter(firstSubiter)
        , pbuffer1(pbuffer1)
        , pbuffer2(pbuffer2)
        , width(width)
        , height(height)
    {
    }

    // Erode (or not) the pixel at the given index
    __host__ __device__
    bool operator()(int index)
    {
        // Goal: use as few if statement as possible!

        int const i = index % width;
        int const j = index / width; // integer division

        Pixel const black = pbuffer1[index];

#ifdef ENABLE_PARALLEL_SHORT_CIRCUIT_WHITE
        if (!black) return false;
#endif

        Pixel const x1    = pbuffer1[indexOf(i + 1, j)];
        Pixel const x2    = pbuffer1[indexOf(i + 1, j - 1)];
        Pixel const x3    = pbuffer1[indexOf(i, j - 1)];
        Pixel const x4    = pbuffer1[indexOf(i - 1, j - 1)];
        Pixel const x5    = pbuffer1[indexOf(i - 1, j)];
        Pixel const x6    = pbuffer1[indexOf(i - 1, j + 1)];
        Pixel const x7    = pbuffer1[indexOf(i, j + 1)];
        Pixel const x8    = pbuffer1[indexOf(i + 1, j + 1)];

        // Xh: Hiditch's crossing number
        int const Xh =
            /* b(1) */ (!x1 & (x2 | x3)) +
            /* b(2) */ (!x3 & (x4 | x5)) +
            /* b(3) */ (!x5 & (x6 | x7)) +
            /* b(4) */ (!x7 & (x8 | x1));

        // G1:
        bool const G1 = (Xh == 1);

        // N1 and N2
        int const N1 = (x2 | x1) + (x4 | x3) + (x6 | x5) + (x8 | x7);
        int const N2 = (x3 | x2) + (x5 | x4) + (x7 | x6) + (x1 | x8);
        int const N = N2 ^ ((N1 ^ N2) & -(N1 < N2)); // min(N1, N2)

        // G2:
        bool const G2 = (2 <= N) & (N <= 3);

        // G3 / G3':
        bool const G3 =
            (firstSubiter and ((x2 | x3 | !x8) & x1) == 0) |
            (!firstSubiter and ((x6 | x7 | !x4) & x5) == 0);

        // All together:
        bool const deleted = black & G1 & G2 & G3;

        // Because WHITE == false we can do as follow:
        pbuffer2[index] = pbuffer1[index] & !deleted;

        return deleted; // 1/true or 0/false
    }

    __host__ __device__
    int indexOf(int i, int j) const { return j * width + i; }
};

bool erodeIter(BWImage<DevicePixels> const& buffer1, BWImage<DevicePixels>& buffer2, bool firstSubiter)
{
    int const w     = buffer1.width;
    int const h     = buffer1.height;
    int const begin = buffer1.indexOf(1, 1);
    int const end   = buffer1.indexOf(w - 1, h - 1);

    // Transformation: take the index as input, modify the second buffer
    //                 and return 1 when the pixel is deleted, 0 otherwise
    // Reduction: addition the number of deleted pixels
    bool eroded = thrust::transform_reduce(
        thrust::make_counting_iterator(begin),
        thrust::make_counting_iterator(end),
        ErodePixelFunctor(firstSubiter, buffer1.pixels.data().get(), buffer2.pixels.data().get(), w, h),
        0,
        thrust::bit_or<bool>()
    );

    // Continue while some pixels were deleted
    return eroded;
}
